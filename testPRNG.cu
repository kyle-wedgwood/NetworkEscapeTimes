
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>

__global__ void init_stuff(hiprandState *state)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   hiprand_init(1337, idx, 0, &state[idx]);
}

__global__ void make_rand(hiprandState *state, float
*randArray)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   randArray[idx] = hiprand_normal(&state[idx]);
}

void SaveData( int npts, float *x, char *filename) {
  FILE *fp = fopen(filename,"w");
  for (int i=0;i<npts;i++) {
    fprintf(fp,"%f\n",x[i]);
  }
  fclose(fp);
}

int main( int argc, char* argv[])
{
   hiprandState *d_state;
   float* randArray;
   float* host_randArray;
   int nThreads = 512;
   int nBlocks  = 1000;

   host_randArray = (float*) malloc( nThreads*nBlocks*sizeof(float));
   hipMalloc (&d_state, nThreads*nBlocks*sizeof(hiprandState));
   hipMalloc( &randArray, nThreads*nBlocks*sizeof(float));

   init_stuff<<<nBlocks, nThreads>>>(d_state);
   make_rand<<<nBlocks, nThreads>>>(d_state, randArray);

   hipMemcpy( host_randArray, randArray, nThreads*nBlocks*sizeof(float),
       hipMemcpyDeviceToHost);

   char filename[] = "testPRNG.dat";
   SaveData(nThreads*nBlocks,host_randArray,filename);

   free(host_randArray);
   hipFree(randArray);
   hipFree(d_state);

   return 0;
}
